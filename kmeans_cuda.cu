#include <stdio.h>
#include <stdlib.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <math.h>
#include <ctype.h>
#include <string.h>
#include <assert.h>
#include <iostream>
#include <hip/hip_runtime.h>

struct KMData {
    int ndata;
    int dim;
    int* features;
    int* assigns;
    int* labels;
    int nlabels;
};

struct KMClust {
    int nclust;
    int dim;
    double* features;
    int* counts;
};

// More in depth CHECK() Macro, gives Cuda error code
// Found this via the internet, I did not write it
#define CHECK(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char* const func, const char* const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    }
}

void get_mem_info() {
    float free_m,total_m,used_m;
    size_t free_t,total_t;
    hipMemGetInfo(&free_t,&total_t);
    free_m =(uint)free_t/1048576.0 ;
    total_m=(uint)total_t/1048576.0;
    used_m=total_m-free_m;
    printf ( "  mem free %d .... %f MB mem total %d....%f MB mem used %f MB\n",free_t,free_m,total_t,total_m,used_m);
}

void transfer() {
    float* dev_tmp;
    CHECK(hipMalloc((void**) &dev_tmp, sizeof(float)));
    float* host_tmp = (float*)malloc(sizeof(float));
    host_tmp[0] = 1.;
    CHECK(hipMemcpy(dev_tmp, host_tmp, sizeof(float), hipMemcpyHostToDevice));
    host_tmp[0] = 9.;
    CHECK(hipMemcpy(host_tmp, dev_tmp, sizeof(float), hipMemcpyDeviceToHost));
    printf("result of memcpy: %f\n", host_tmp[0]);

    hipFree(dev_tmp);
    free(host_tmp);
}

int filestats(char *filename, ssize_t *tot_tokens, ssize_t *tot_lines){
// Sets number of lines and total number of whitespace separated
// tokens in the file. Returns -1 if file can't be opened, 0 on
// success.
//
// EXAMPLE: int ret = filestats("digits_all_1e1.txt", &toks, &lines);
// toks  is now 7860 : 10 lines with 786 tokens per line, label + ":" + 28x28 pixels
// lines is now 10   : there are 10 lines in the file
    FILE *fin = fopen(filename,"r");
    if(fin == NULL){
        printf("Failed to open file '%s'\n",filename);
        return -1;
    }

    ssize_t ntokens=0, nlines=0, column=0;
    int intoken=0, token;
    while((token = fgetc(fin)) != EOF){
        if(token == '\n'){          // reached end of line
        column = 0;
        nlines++;
        }
        else{
            column++;
        }
        if(isspace(token) && intoken==1){ // token to whitespace
            intoken = 0;
        }
        else if(!isspace(token) && intoken==0){ // whitespace to token
            intoken = 1;
            ntokens++;
        }
    }
    if(column != 0){              // didn't end with a newline
        nlines++;                   // add a line on to the count
    }
    *tot_tokens = ntokens;
    *tot_lines = nlines;
    fclose(fin);
  // printf("DBG: tokens: %lu\n",ntokens);
  // printf("DBG: lines: %lu\n",nlines);
    return 0;
}


struct KMData * kmdata_load(struct KMData *data, char* datafile) {
    ssize_t tot_tokens, tot_lines;
    data->ndata = 0;
    int stat = filestats(datafile, &tot_tokens, &tot_lines);
    if (stat == -1) {
        printf("filestats return stat: %d", stat);
        return data;
    }

    FILE *fin = fopen(datafile, "r");
    if (fin == NULL) { printf("error opening file\n"); }
    data->ndata = tot_lines;
    int line_size = (tot_tokens / tot_lines) - 2;
    int max_label = 0;
    data->labels = (int*)malloc(tot_lines * sizeof(int));
    data->features = (int*)malloc(tot_lines * line_size * sizeof(int));
    char line[3142];
    int row = 0;
    int c;
    while (fgets(line, 3142*sizeof(char), fin) != NULL) {
        char* token = strtok(line, " ");
        data->labels[row] = atoi(token);
        max_label = (atoi(token) > max_label) ? atoi(token) : max_label;
        token = strtok(NULL, " ");
        token = strtok(NULL, " ");
        c = 1;
        while (token != NULL) {
            data->features[row * line_size + c - 1] = atoi(token);
            token = strtok(NULL, " ");
            c++;
        }
        row++;
    }
    data->assigns = (int*)malloc(tot_lines * sizeof(int));
    data->dim = line_size;
    data->nlabels = max_label + 1;
    fclose(fin);
    return data;
}

struct KMClust * kmclust_new(struct KMClust *clust, int nclust, int dim) {
    clust->nclust = nclust;
    clust->dim = dim;
    clust->features = (double*)malloc(nclust * dim * sizeof(double*));
    clust->counts = (int*)malloc(nclust * sizeof(int));
    return clust;
}

void save_pgm_files(struct KMClust *clust, char* savedir) {
    int dim_root = (int)(sqrt((clust->dim)));
    if (clust->dim % dim_root == 0) {
        double maxfeats = 0.0;
        int dim = clust->dim;
        int nclust = clust->nclust;
        for (int i=0; i<dim; i++) {
            for (int j=0; j<nclust; j++) {
                maxfeats = (clust->features[j * clust->dim + i] > maxfeats) ? clust->features[j * clust->dim + i] : maxfeats;
            }
        }
        
        for (int c=0; c<nclust; c++) {
            char cent[10];
            char* pgm = ".pgm";
            char numbuf[11];

            if (c < 10) {
                sprintf(cent, "/cent_000");
            }
            else if ((100 > c) && (c >= 10)) {
                sprintf(cent, "/cent_00");
            }
            else {
                sprintf(cent, "/cent_0");
            }

            sprintf(numbuf, "%d", c);
            char outfile[128];
            sprintf(outfile, "%s%s%s%s", savedir, cent, numbuf, pgm);
            FILE *fout = fopen(outfile, "w+");
            if (fout == NULL) {
                printf("error creating file: %s\n", outfile);
                return;
            }

            fprintf(fout, "P2\n%d %d\n%3.0f\n", dim_root, dim_root, maxfeats);
            for (int d=0; d<dim; d++) {
                if ((d > 0) && (d % dim_root == 0)) {
                    fprintf(fout, "\n");
                }
                fprintf(fout, "%3.0f ", clust->features[c * clust->dim + d]);
            }

            fprintf(fout, "\n");
            fclose(fout);
        }
    }
    printf("Saving cluster centers to %s/cent_0000.pgm ...\n", savedir);
}

void create_cuda_clust_data(KMClust* clust, KMData* data, KMClust* dev_clust, KMData* dev_data) {    


    dev_clust->nclust = clust->nclust;
    dev_clust->dim = clust->dim;
    CHECK(hipMalloc((void**) &dev_clust->features, clust->nclust * clust->dim * sizeof(double)));
    CHECK(hipMemcpy(dev_clust->features, clust->features, clust->dim * clust->nclust * sizeof(double), hipMemcpyHostToDevice));
    
    CHECK(hipMalloc((void**) &dev_clust->counts, clust->nclust * sizeof(int)));
    CHECK(hipMemcpy(dev_clust->counts, clust->counts, clust->nclust * sizeof(int), hipMemcpyHostToDevice));
    

    dev_data->ndata = data->ndata;

    CHECK(hipMalloc((void**) &dev_data->labels, data->ndata * sizeof(int)));
    CHECK(hipMemcpy(dev_data->labels, data->labels, data->ndata * sizeof(int), hipMemcpyHostToDevice));

    CHECK(hipMalloc((void**) &dev_data->features, data->ndata * data->dim * sizeof(int)));
    CHECK(hipMemcpy(dev_data->features, data->features, data->dim * data->ndata * sizeof(int), hipMemcpyHostToDevice));

    CHECK(hipMalloc((void**) &dev_data->assigns, data->ndata * sizeof(int)));
    CHECK(hipMemcpy(dev_data->assigns, data->assigns, data->ndata * sizeof(int), hipMemcpyHostToDevice));
    dev_data->dim = data->dim;
    dev_data->nlabels = data->nlabels;
}

void sync_device_host(KMClust* clust, KMClust* ptr_clust, KMData* data, KMData* ptr_data) {    
    CHECK(hipMemcpy(clust->counts, ptr_clust->counts, clust->nclust * sizeof(int), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(data->assigns, ptr_data->assigns, data->ndata * sizeof(int), hipMemcpyDeviceToHost));
}

__global__ void new_cluster_centers(KMClust* clust, KMData* data) {
    long idx = threadIdx.x;
    if (idx < 784) {
        int c;

        for (int i=0; i<data->ndata; i++) {
            c = data->assigns[i];
            clust->features[c * clust->dim + idx] += data->features[i * clust->dim + idx];
        }

        for (int i=0; i<clust->nclust; i++) {
            if (clust->counts[i] > 0) {
                clust->features[i * clust->dim + idx] = clust->features[i * clust->dim + idx] / clust->counts[i];
            }
        }
    }
    __syncthreads();

}

__global__ void new_assignments(KMClust* clust, KMData* data, int numThreads, int* nchanges) {
    long idx = threadIdx.x + blockIdx.x * numThreads;
    if (idx < clust->nclust) {
        clust->counts[idx] = 0;
    }
    if (idx < data->ndata) {
        int best_clust = -1;
        float best_distsq = INFINITY;
        float distsq;
        for (int c=0; c<clust->nclust; c++) {
            distsq = 0.0;
            for (int d=0; d<clust->dim; d++) {
                float diff = data->features[idx * clust->dim + d] - clust->features[c * clust->dim + d];
                distsq += diff * diff;
            }

            if (distsq < best_distsq) {
                best_clust = c;
                best_distsq = distsq;
            }
        }
        atomicAdd(&clust->counts[best_clust], 1);
        if (best_clust != data->assigns[idx]) {
            atomicAdd(&nchanges[0], 1);
            data->assigns[idx] = best_clust;
        }
    }


    __syncthreads();
}

int main(int argc, char *argv[]) {
    if (argc < 3) {
        printf("usage: kmeans.exe <datafile> <nclust> [savedir] [maxiter]\n");
        return 0;
    }

    char* datafile = argv[1];
    int nclust = atoi(argv[2]);
    char* savedir = ".";
    int MAXITER = 100;

    if (argc > 3) {
        savedir = argv[3];
        mkdir(savedir, 0777);
        // mkdir(savedir, 0700);
    }

    if (argc > 4) {
        MAXITER = atoi(argv[4]);
    }

    printf("datafile: %s\nnclust: %d\nsavedir: %s\n", datafile, nclust, savedir);
    struct KMData *data = (struct KMData*)malloc(sizeof(struct KMData));
    data = kmdata_load(data, datafile);
    struct KMClust *clust = (struct KMClust*)malloc(sizeof(struct KMClust));
    kmclust_new(clust, nclust, data->dim);

    printf("ndata: %d\ndim: %d\n\n", data->ndata, data->dim);

    int c;
    for (int i=0; i<data->ndata; i++) {
        c = i % clust->nclust;
        data->assigns[i] = c;
    }
    
    double icount;
    int extra;
    for (int i=0; i<clust->nclust; i++) {
        icount = data->ndata / clust->nclust;
        extra = 0;
        if (i < data->ndata % clust->nclust) {
            extra = 1;
        }
        clust->counts[i] = icount + extra;
    }

    struct KMClust* ptr_clust = (struct KMClust*)malloc(sizeof(KMClust));
    struct KMData* ptr_data = (struct KMData*)malloc(sizeof(KMData));
    create_cuda_clust_data(clust, data, ptr_clust, ptr_data);

    struct KMClust* dev_clust;
    struct KMData* dev_data;

    hipMalloc((void**) &dev_clust, sizeof(KMClust));
    hipMalloc((void**) &dev_data, sizeof(KMData));

    hipMemcpy(dev_clust, ptr_clust, sizeof(KMClust), hipMemcpyHostToDevice);
    hipMemcpy(dev_data, ptr_data, sizeof(KMData), hipMemcpyHostToDevice);




    int curiter = 1;
    int nchanges = data->ndata;

    int numThreads = 1024;
    int numBlocks = (data->ndata / numThreads) + 1;

    int* dev_nchanges;
    hipMalloc((void**) &dev_nchanges, sizeof(int));

    printf("==CLUSTERING: MAXITER %d==\n", MAXITER);
    printf("ITER NCHANGE CLUST_COUNTS\n");

    int* iter;

    while (nchanges > 0 && curiter <= MAXITER) {

        // Set cluster features to 0
        CHECK(hipMemset(ptr_clust->features, 0., clust->dim * clust->nclust * sizeof(double)));

        // Calculate new cluster centers
        new_cluster_centers<<<1, 784>>>(dev_clust, dev_data);

        // Ensure all threads finish and sync the GPU assigns and counts
        CHECK(hipDeviceSynchronize());
        sync_device_host(clust, ptr_clust, data, ptr_data);
        
        nchanges = 0;
        CHECK(hipMemset(dev_nchanges, 0, sizeof(int)));

        // Calculate new assignments
        new_assignments<<<numBlocks, numThreads>>>(dev_clust, dev_data, numThreads, dev_nchanges);
        CHECK(hipDeviceSynchronize());
        CHECK(hipMemcpy(&nchanges, dev_nchanges, sizeof(int), hipMemcpyDeviceToHost));

        sync_device_host(clust, ptr_clust, data, ptr_data);

        printf(" %d:%6d |", curiter, nchanges);
        for (int i=0; i<clust->nclust; i++) {
            printf("%5d", clust->counts[i]);
        }
        printf("\n");
        curiter++;
    }

    printf("CONVERGED: after %d iterations\n", curiter);

    int confusion[data->nlabels][nclust];
    for (int i=0; i<data->nlabels; i++){
        for (int j=0; j<nclust; j++) {
            confusion[i][j] = 0;
        }
    }
    for (int i=0; i<data->ndata; i++) {
        confusion[data->labels[i]][data->assigns[i]] += 1;
    }

    printf("\n==CONFUSION MATRIX + COUNTS==\n");
    printf("LABEL \\ CLUST\n");
    printf("   ");
    // printf("%5s"," ");
    for (int j=0; j<clust->nclust; j++) {
        printf("%5d", j);
    }
    printf("   TOT\n");

    int tot;
    for (int i=0; i<data->nlabels; i++) {
        printf(" %d:", i);
        tot = 0;
        for (int j=0; j<clust->nclust; j++) {
            printf("%5d", confusion[i][j]);
            tot += confusion[i][j];
        }
        printf("%5d\n", tot);
    }

    printf("TOT");
    tot = 0;

    for (int c=0; c<clust->nclust; c++) {
        printf("%5d", clust->counts[c]);
        tot += clust->counts[c];
    }

    printf(" %d \n", tot);

    char* labels = "/labels.txt";
    char outfile[128];
    sprintf(outfile, "%s%s", savedir, labels);
    FILE *fout = fopen(outfile, "w+");
    if (fout == NULL) {
        printf("error creating file: %s\n", outfile);
        return 1;
    }
    for (int i=0; i<data->ndata; i++) {
        fprintf(fout, "%2d %2d\n",data->labels[i], data->assigns[i]);
    }
    fclose(fout);
    printf("Saving cluster labels to file %s/labels.txt\n", savedir);
    CHECK(hipMemcpy(clust->features, ptr_clust->features, clust->nclust * clust->dim * sizeof(double), hipMemcpyDeviceToHost));
    save_pgm_files(clust, savedir);
    // Freeing stuff

    free(data->features);
    free(data->assigns);
    free(data->labels);
    hipFree(ptr_data->features);

    hipFree(ptr_data->assigns);
    hipFree(ptr_data->labels);

    free(clust->features);
    free(clust->counts);
    free(clust);
    free(data);

    hipFree(ptr_clust->features);
    hipFree(ptr_clust->counts);

    free(ptr_clust);
    free(ptr_data);
    hipFree(dev_clust);
    hipFree(dev_data);
    hipFree(dev_nchanges);
    return 0;
}